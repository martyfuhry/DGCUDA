#include "hip/hip_runtime.h"
#include "../main.cu"

/* shallowwater.cu
 *
 */

#define G 9.8

__device__ void get_velocity(double *, double, double, double);

/* size of the system */
int local_N = 3;

/***********************
 *
 * INITIAL CONDITIONS
 *
 ***********************/

__device__ void evalU0(double *U, double *V, int i) {
    int j;
    double X[2];
    double u0[3];

    U[0] = 0.;
    U[1] = 0.;
    U[2] = 0.;

    for (j = 0; j < n_quad; j++) {

        // get the 2d point on the mesh
        get_coordinates_2d(X, V, j);

        // evaluate U0 here
        U0(u0, X[0], X[1]);

        // evaluate U at the integration point
        U[0] += w[j] * u0[0] * basis[i * n_quad + j];
        U[1] += w[j] * u0[1] * basis[i * n_quad + j];
        U[2] += w[j] * u0[2] * basis[i * n_quad + j];
    }
}

__device__ double eval_c(double *U) {
    return sqrt(U[0]*G);
}

__device__ bool is_physical(double *U) {
    return U[0] >= 0.;
}

/* check physical
 *
 * if U isn't physical, replace the solution with the constant average value
 */
__device__ void check_physical(double *C_global, double *C, double *U, int idx) {
    int i;

    // check to see if U is physical
    if (!is_physical(U)) {
        // set C[1] to C[n_p] to zero
        for (i = 1; i < n_p; i++) {
            C_global[num_elem * n_p * 0 + i * num_elem + idx] = 0.;
            C_global[num_elem * n_p * 1 + i * num_elem + idx] = 0.;
            C_global[num_elem * n_p * 2 + i * num_elem + idx] = 0.;
            C_global[num_elem * n_p * 3 + i * num_elem + idx] = 0.;

            C[n_p * 0 + i] = 0.;
            C[n_p * 1 + i] = 0.;
            C[n_p * 2 + i] = 0.;
            C[n_p * 3 + i] = 0.;
        }

        // rebuild the solution as simply the average value
        U[0] = C[n_p * 0 + 0] * basis[0];
        U[1] = C[n_p * 1 + 0] * basis[0];
        U[2] = C[n_p * 2 + 0] * basis[0];
        U[3] = C[n_p * 3 + 0] * basis[0];
    }
}


/***********************
 *
 * SHALLOWWATER FLUX
 *
 ***********************/
 /*
  * sets the flux for advection
 */
__device__ void eval_flux(double *U, double *flux_x, double *flux_y,
                          double *V, double t, int j, int left_side) {

    double h, uh, vh;

    h  = U[0];
    uh = U[1];
    vh = U[2];

    // flux_1 
    flux_x[0] = uh;
    flux_x[1] = uh*uh/h + 0.5*G*h*h;
    flux_x[2] = uh*vh/h;

    // flux_2
    flux_y[0] = vh;
    flux_y[1] = uh*vh/h;
    flux_y[2] = vh*vh/h + 0.5*G*h*h;
}

/***********************
 *
 * RIEMAN SOLVER
 *
 ***********************/
/* finds the max absolute value of the jacobian for F(u).
 */
__device__ double eval_lambda(double *U_left, double *U_right,
                              double *V,      double t,
                              double nx,      double ny,
                              int j, int left_side) {
                              
    double s_left, s_right;
    double c_left, c_right;
    double u_left, v_left;
    double u_right, v_right;
    double left_max, right_max;

    // get c for both sides
    c_left  = eval_c(U_left);
    c_right = eval_c(U_right);

    // find the speeds on each side
    u_left  = U_left[1] / U_left[0];
    v_left  = U_left[2] / U_left[0];
    u_right = U_right[1] / U_right[0];
    v_right = U_right[2] / U_right[0];
    s_left  = nx * u_left  + ny * v_left;
    s_right = nx * u_right + ny * v_right; 
    
    // if speed is positive, want s + c, else s - c
    if (s_left > 0.) {
        left_max = s_left + c_left;
    } else {
        left_max = -s_left + c_left;
    }

    // if speed is positive, want s + c, else s - c
    if (s_right > 0.) {
        right_max = s_right + c_right;
    } else {
        right_max = -s_right + c_right;
    }

    // return the max absolute value of | s +- c |
    if (abs(left_max) > abs(right_max)) {
        return abs(left_max);
    } else { 
        return abs(right_max);
    }
}

/* local lax-friedrichs riemann solver
 */
__device__ void riemann_solver(double *F_n, double *U_left, double *U_right,
                               double *V, double t,
                               double nx, double ny,
                               int j, int left_side) {
    int n;

    double flux_x_l[3], flux_x_r[3];
    double flux_y_l[3], flux_y_r[3];

    // calculate the left and right fluxes
    eval_flux(U_left, flux_x_l, flux_y_l, V, t, j, left_side);
    eval_flux(U_right, flux_x_r, flux_y_r, V, t, j, left_side);

    double lambda = eval_lambda(U_left, U_right, V, t, nx, ny, j, left_side);

    // calculate the riemann problem at this integration point
    for (n = 0; n < N; n++) {
        F_n[n] = 0.5 * ((flux_x_l[n] + flux_x_r[n]) * nx + (flux_y_l[n] + flux_y_r[n]) * ny 
                    + lambda * (U_left[n] - U_right[n]));
    }
}

/***********************
 *
 * CFL CONDITION
 *
 ***********************/
/* global lambda evaluation
 *
 * computes the max eigenvalue of |u + c|, |u|, |u - c|.
 */
__global__ void eval_global_lambda(double *C, double *lambda, 
                                   double *V1x, double *V1y,
                                   double *V2x, double *V2y,
                                   double *V3x, double *V3y,
                                   double t) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elem) { 
        double c, s;

        double U[3];
        // get cell averages
        U[0] = C[num_elem * n_p * 0 + idx] * basis[0];
        U[1] = C[num_elem * n_p * 1 + idx] * basis[0];
        U[2] = C[num_elem * n_p * 2 + idx] * basis[0];

        // evaluate c
        c = eval_c(U);

        // speed of the wave
        s = sqrt(U[1]*U[1] + U[2]*U[2])/U[0];

        // return the max eigenvalue
        if (s > 0) {
            lambda[idx] = s + c;
        } else {
            lambda[idx] = -s + c;
        }
    }
}
