#include "hip/hip_runtime.h"
/* time_integrator.cu
 *
 * time integration functions.
 */
#ifndef TIMEINTEGRATOR_H_GUARD
#define TIMEINTEGRATOR_H_GUARD
void checkCudaError(const char*);
#endif

extern int local_N;
extern int limiter;

void write_U(int, int, int);

/***********************
 * ASSEMBLE RHS FUNCTIONS
 ***********************/

/* right hand side
 *
 * computes the sum of the quadrature and the riemann flux for the 
 * coefficients for each element
 * THREADS: num_elem
 */
__global__ void eval_rhs(double *c, double *rhs_volume, double *rhs_surface_left, double *rhs_surface_right, 
                         int *elem_s1, int *elem_s2, int *elem_s3,
                         int *left_elem, double *J, double dt) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    double j;
    int i, s1_idx, s2_idx, s3_idx;
    int n;

    if (idx < num_elem) {

        // set to 0
        for (i = 0; i < n_p; i++) { 
            for (n = 0; n < N; n++) {
                c[num_elem * n_p * n + i * num_elem + idx] = 0.;
            }
        }

        // read jacobian determinant
        j = J[idx];

        // get the indicies for the riemann contributions for this element
        s1_idx = elem_s1[idx];
        s2_idx = elem_s2[idx];
        s3_idx = elem_s3[idx];

        // add volume integral
        for (i = 0; i < n_p; i++) {
            for (n = 0; n < N; n++) {
                c[num_elem * n_p * n + i * num_elem + idx] += dt / j * rhs_volume[num_elem * n_p * n + i * num_elem + idx];
            }
        }

        // for the first edge, add either left or right surface integral
        if (idx == left_elem[s1_idx]) {
            for (i = 0; i < n_p; i++) {
                for (n = 0; n < N; n++) {
                    c[num_elem * n_p * n + i * num_elem + idx] += dt / j * rhs_surface_left[num_sides * n_p * n + i * num_sides + s1_idx];
                }
            }
        } else {
            for (i = 0; i < n_p; i++) {
                for (n = 0; n < N; n++) {
                    c[num_elem * n_p * n + i * num_elem + idx] += dt / j * rhs_surface_right[num_sides * n_p * n + i * num_sides + s1_idx];
                }
            }
        }
        // for the second edge, add either left or right surface integral
        if (idx == left_elem[s2_idx]) {
            for (i = 0; i < n_p; i++) {
                for (n = 0; n < N; n++) {
                    c[num_elem * n_p * n + i * num_elem + idx] += dt / j * rhs_surface_left[num_sides * n_p * n + i * num_sides + s2_idx];
                }
            }
        } else {
            for (i = 0; i < n_p; i++) {
                for (n = 0; n < N; n++) {
                    c[num_elem * n_p * n + i * num_elem + idx] += dt / j * rhs_surface_right[num_sides * n_p * n + i * num_sides + s2_idx];
                }
            }
        }
        // for the third edge, add either left or right surface integral
        if (idx == left_elem[s3_idx]) {
            for (i = 0; i < n_p; i++) {
                for (n = 0; n < N; n++) {
                    c[num_elem * n_p * n + i * num_elem + idx] += dt / j * rhs_surface_left[num_sides * n_p * n + i * num_sides + s3_idx];
                }
            }
        } else {
            for (i = 0; i < n_p; i++) {
                for (n = 0; n < N; n++) {
                    c[num_elem * n_p * n + i * num_elem + idx] += dt / j * rhs_surface_right[num_sides * n_p * n + i * num_sides + s3_idx];
                }
            }
        }
    }
}

/* tempstorage for RK
 * 
 * I need to store u + alpha * k_i into some temporary variable called kstar
 */
__global__ void rk_tempstorage(double *c, double *kstar, double*k, double alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < N * n_p * num_elem) {
        kstar[idx] = c[idx] + alpha * k[idx];
    }
}

/***********************
 * RK4 
 ***********************/

/* rk4
 *
 * computes the runge-kutta solution 
 * u_n+1 = u_n + k1/6 + k2/3 + k3/3 + k4/6
 */
__global__ void rk4(double *c, double *k1, double *k2, double *k3, double *k4) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < N * n_p * num_elem) {
        c[idx] += k1[idx]/6. + k2[idx]/3. + k3[idx]/3. + k4[idx]/6.;
    }
}



/* time integrate rk4
 *
 * uses fourth order runge-kutta time integration to solve the RHS.
 * returns the final time this runs to.
 */
double time_integrate_rk4(int local_num_elem, int local_num_sides, 
                          int local_n, int local_n_p,
                          double endtime, int total_timesteps, double min_r, 
                          int verbose, int convergence, int video, double tol) {
    int n_threads = 256;
    int i, vidnum;
    double dt, t;
    double *c;
    double conv;

    if (convergence) {
        c = (double *) malloc(local_num_elem * local_n_p * local_N * sizeof(double));
    }
    double *max_lambda = (double *) malloc(local_num_elem * sizeof(double));
    double max_l;

    int n_blocks_elem  = (local_num_elem  / n_threads) 
                       + ((local_num_elem  % n_threads) ? 1 : 0);
    int n_blocks_sides = (local_num_sides / n_threads) 
                       + ((local_num_sides % n_threads) ? 1 : 0);
    int n_blocks_rk    = ((local_N * local_n_p * local_num_elem) / n_threads) 
                       + (((local_N * local_n_p * local_num_elem) % n_threads) ? 1 : 0);

    t = 0;
    int timestep = 0;

    // limit before stage 1
    if (limiter) {
        limit_c<<<n_blocks_elem, n_threads>>>(d_c, d_elem_s1, d_elem_s2, d_elem_s3,
                                              d_left_elem, d_right_elem);
                                              
        hipDeviceSynchronize();
    }

    // write initial conditions if video
    vidnum = 0;
    if (video > 0) {
        if (timestep % video == 0) {
            write_U(local_num_elem, vidnum, total_timesteps);
            hipDeviceSynchronize();
            vidnum++;
        }
    }

    conv = 1.;
    printf("Computing...\n");
    while (t < endtime || (timestep < total_timesteps && total_timesteps != -1) || (convergence && conv > tol)) {
        // compute all the lambda values over each cell
        eval_global_lambda<<<n_blocks_elem, n_threads>>>(d_c, d_lambda, 
                                                         d_V1x, d_V1y,
                                                         d_V2x, d_V2y,
                                                         d_V3x, d_V3y,
                                                         t);

        // grab all the lambdas off the GPU and find the min one
        hipMemcpy(max_lambda, d_lambda, local_num_elem * sizeof(double), hipMemcpyDeviceToHost);
        max_l = max_lambda[0];
        for (i = 0; i < local_num_elem; i++) {
            max_l = (max_lambda[i] > max_l) ? max_lambda[i] : max_l;
        }
        checkCudaError("error after eval_lambda");

        timestep++;

        //cfl condition
        dt = 0.7 * min_r / max_l /  (2. * local_n + 1.);

        // panic
        if (isnan(dt)) {
            printf("Error: dt is NaN. Dumping...\n");
            return t;
        }

        // get next timestep
        if (t + dt > endtime && total_timesteps == -1 && convergence != 1) {
            dt = endtime - t;
            t = endtime;
        } else {
            t += dt;
        }

        if (verbose == 1) {
            printf("(%i) t = %lf, dt = %lf, max_l = %lf\n", timestep, t, dt, max_l);
        } else if (convergence == 1)  {
            printf("\r(%i) t = %lf, convergence = %.015lf", timestep, t, conv);
        }
        else {
            printf("\r(%i) t = %lf", timestep, t);
        }

        // stage 1
        hipDeviceSynchronize();
        checkCudaError("error before stage 1: eval_surface");
        eval_surface<<<n_blocks_sides, n_threads>>>
                      (d_c, d_rhs_surface_left, d_rhs_surface_right, 
                       d_s_length, 
                       d_V1x, d_V1y,
                       d_V2x, d_V2y,
                       d_V3x, d_V3y,
                       d_left_elem, d_right_elem,
                       d_left_side_number, d_right_side_number,
                       d_Nx, d_Ny, t);

        checkCudaError("error after stage 1: eval_surface");

        eval_volume<<<n_blocks_elem, n_threads>>>
                        (d_c, d_rhs_volume, 
                         d_xr, d_yr, d_xs, d_ys,
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         t);
        hipDeviceSynchronize();

        checkCudaError("error after stage 1: eval_volume");

        eval_rhs<<<n_blocks_elem, n_threads>>>(d_k1, d_rhs_volume, d_rhs_surface_left, d_rhs_surface_right, 
                                              d_elem_s1, d_elem_s2, d_elem_s3, 
                                              d_left_elem, d_J, dt);
        hipDeviceSynchronize();

        if (limiter) {
            limit_c<<<n_blocks_elem, n_threads>>>(d_k1, d_elem_s1, d_elem_s2, d_elem_s3,
                                                  d_left_elem, d_right_elem);
                                                  
            hipDeviceSynchronize();
        }

        rk_tempstorage<<<n_blocks_rk, n_threads>>>(d_c, d_k2, d_k1, 0.5);
        hipDeviceSynchronize();
        checkCudaError("error after stage 1.");

        // stage 2
        eval_surface<<<n_blocks_sides, n_threads>>>
                        (d_k2, d_rhs_surface_left, d_rhs_surface_right, 
                         d_s_length, 
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         d_left_elem, d_right_elem,
                         d_left_side_number, d_right_side_number,
                         d_Nx, d_Ny, 
                         t + 0.5*dt);

        eval_volume<<<n_blocks_elem, n_threads>>>
                        (d_k2, d_rhs_volume, 
                         d_xr, d_yr, d_xs, d_ys,
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         t + 0.5*dt);
        hipDeviceSynchronize();

        eval_rhs<<<n_blocks_elem, n_threads>>>(d_k2, d_rhs_volume, d_rhs_surface_left, d_rhs_surface_right,
                                              d_elem_s1, d_elem_s2, d_elem_s3, 
                                              d_left_elem, d_J, dt);
        hipDeviceSynchronize();

        if (limiter) {
            limit_c<<<n_blocks_elem, n_threads>>>(d_k2, d_elem_s1, d_elem_s2, d_elem_s3,
                                                  d_left_elem, d_right_elem);
            hipDeviceSynchronize();
        }


        rk_tempstorage<<<n_blocks_rk, n_threads>>>(d_c, d_k3, d_k2, 0.5);
        hipDeviceSynchronize();

        checkCudaError("error after stage 2.");

        // stage 3
        eval_surface<<<n_blocks_sides, n_threads>>>
                        (d_k3, d_rhs_surface_left, d_rhs_surface_right, 
                         d_s_length, 
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         d_left_elem, d_right_elem,
                         d_left_side_number, d_right_side_number,
                         d_Nx, d_Ny, 
                         t + 0.5*dt);

        eval_volume<<<n_blocks_elem, n_threads>>>
                        (d_k3, d_rhs_volume, 
                         d_xr, d_yr, d_xs, d_ys,
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         t + 0.5*dt);
        hipDeviceSynchronize();

        eval_rhs<<<n_blocks_elem, n_threads>>>(d_k3, d_rhs_volume, d_rhs_surface_left, d_rhs_surface_right, 
                                              d_elem_s1, d_elem_s2, d_elem_s3, 
                                              d_left_elem, d_J, dt);
        hipDeviceSynchronize();

        if (limiter) {
            limit_c<<<n_blocks_elem, n_threads>>>(d_k3, d_elem_s1, d_elem_s2, d_elem_s3,
                                                  d_left_elem, d_right_elem);
            hipDeviceSynchronize();
        }

        rk_tempstorage<<<n_blocks_rk, n_threads>>>(d_c, d_k4, d_k3, 1.0);
        hipDeviceSynchronize();

        checkCudaError("error after stage 3.");

        // stage 4
        eval_surface<<<n_blocks_sides, n_threads>>>
                        (d_k4, d_rhs_surface_left, d_rhs_surface_right, 
                         d_s_length, 
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         d_left_elem, d_right_elem,
                         d_left_side_number, d_right_side_number,
                         d_Nx, d_Ny, 
                         t + dt);

        eval_volume<<<n_blocks_elem, n_threads>>>
                        (d_k4, d_rhs_volume, 
                         d_xr, d_yr, d_xs, d_ys,
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         t + dt);
        hipDeviceSynchronize();

        eval_rhs<<<n_blocks_elem, n_threads>>>(d_k4, d_rhs_volume, d_rhs_surface_left, d_rhs_surface_right, 
                                              d_elem_s1, d_elem_s2, d_elem_s3, 
                                              d_left_elem, d_J, dt);
        hipDeviceSynchronize();

        if (limiter) {
            limit_c<<<n_blocks_elem, n_threads>>>(d_k4, d_elem_s1, d_elem_s2, d_elem_s3,
                                                  d_left_elem, d_right_elem);
            hipDeviceSynchronize();
        }

        checkCudaError("error after stage 4.");
        
        // final stage
        rk4<<<n_blocks_rk, n_threads>>>(d_c, d_k1, d_k2, d_k3, d_k4);
        hipDeviceSynchronize();

        if (limiter) {
            limit_c<<<n_blocks_elem, n_threads>>>(d_c, d_elem_s1, d_elem_s2, d_elem_s3,
                                                  d_left_elem, d_right_elem);
            hipDeviceSynchronize();
        }


        // check the convergence
        if (convergence && timestep > 0) {
            check_convergence<<<n_blocks_rk, n_threads>>>(d_c_prev, d_c);
            hipMemcpy(c, d_c_prev, local_num_elem * local_N * sizeof(double), hipMemcpyDeviceToHost);

            conv = c[0];
            for (i = 1; i < local_num_elem * local_N; i++) {
                if (c[i] > conv) {
                    conv = c[i];
                }
            }
        }
        if (convergence) {
            hipMemcpy(d_c_prev, d_c, local_num_elem * local_n_p * local_N * sizeof(double), hipMemcpyDeviceToDevice);
        }


        hipDeviceSynchronize();
        checkCudaError("error after final stage.");

        // evaluate and write the solution
        if (video > 0) {
            if (timestep % video == 0) {
                write_U(local_num_elem, vidnum, total_timesteps);
                hipDeviceSynchronize();
                vidnum++;
            }
        }
    }

    printf("\n");
    free(max_lambda);
    if (convergence) {
        free(c);
    }
    return t;
}


/***********************
 * RK2 
 ***********************/

/* tempstorage for RK2
 * 
 * I need to store u + alpha * k_i into some temporary variable called k*.
 */
__global__ void rk2_tempstorage(double *c, double *kstar, double*k, double alpha, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < N * n_p * num_elem) {
        kstar[idx] = c[idx] + alpha * k[idx];
    }
}

/* rk2
 *
 * computes the runge-kutta solution 
 * u_n+1 = u_n + k1/6 + k2/3 + k3/3 + k4/6
 */
__global__ void rk2(double *c, double *k) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < N * n_p * num_elem) {
        c[idx] += k[idx];
    }
}

double time_integrate_rk2(int local_num_elem, int local_num_sides, 
                          int local_n, int local_n_p,
                          double endtime, int total_timesteps, double min_r, 
                          int verbose, int convergence, int video, double tol) {
    int n_threads = 512;
    int i, timestep;
    double *c;
    double dt, t;

    double *max_lambda = (double *) malloc(local_num_elem * sizeof(double));
    double max_l;
    double conv;

    int n_blocks_elem  = (local_num_elem  / n_threads) 
                       + ((local_num_elem  % n_threads) ? 1 : 0);
    int n_blocks_sides = (local_num_sides / n_threads) 
                       + ((local_num_sides % n_threads) ? 1 : 0);
    int n_blocks_rk    = ((local_N * local_n_p * local_num_elem) / n_threads) 
                       + (((local_N * local_n_p * local_num_elem) % n_threads) ? 1 : 0);

    if (convergence) {
        c = (double *) malloc(local_num_elem * local_n_p * local_N * sizeof(double));
    }

    t = 0;
    timestep = 0;

    conv = 1;
    printf("Computing...\n");
    while (t < endtime || (timestep < total_timesteps && total_timesteps != -1)) {
        // compute all the lambda values over each cell
        eval_global_lambda<<<n_blocks_elem, n_threads>>>(d_c, d_lambda, 
                                                         d_V1x, d_V1y,
                                                         d_V2x, d_V2y,
                                                         d_V3x, d_V3y, t);

        // just grab all the lambdas and sort them since there are so few of them
        hipMemcpy(max_lambda, d_lambda, local_num_elem * sizeof(double), hipMemcpyDeviceToHost);
        max_l = max_lambda[0];
        for (i = 0; i < local_num_elem; i++) {
            max_l = (max_lambda[i] > max_l) ? max_lambda[i] : max_l;
        }

        timestep++;

        // cfl condition
        dt = 0.7 * min_r / max_l /  (2. * local_n + 1.);

        // panic
        if (isnan(dt)) {
            printf("Error: dt is NaN. Dumping...\n");
            return t;
        }

        if (t + dt > endtime && total_timesteps == -1) {
            dt = endtime - t;
            t = endtime;
        } else {
            t += dt;
        }

        if (verbose == 1) {
            printf("t = %lf, dt = %lf, max_l = %lf\n", t, dt, max_l);
        } else {
            printf("\rt = %lf", t);
        }

        // stage 1
        hipDeviceSynchronize();
        checkCudaError("error before stage 1: eval_surface");
        eval_surface<<<n_blocks_sides, n_threads>>>
                        (d_c, d_rhs_surface_left, d_rhs_surface_right, 
                         d_s_length, 
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         d_left_elem, d_right_elem,
                         d_left_side_number, d_right_side_number,
                         d_Nx, d_Ny, t);

        hipDeviceSynchronize();
        checkCudaError("error after stage 1: eval_surface");

        eval_volume<<<n_blocks_elem, n_threads>>>
                        (d_c, d_rhs_volume, 
                         d_xr, d_yr, d_xs, d_ys,
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y, t);
        hipDeviceSynchronize();

        checkCudaError("error after stage 1: eval_volume");

        eval_rhs<<<n_blocks_elem, n_threads>>>(d_k1, d_rhs_volume, d_rhs_surface_left, d_rhs_surface_right, 
                                              d_elem_s1, d_elem_s2, d_elem_s3, 
                                              d_left_elem, d_J, dt);
        hipDeviceSynchronize();

        if (limiter) {
            limit_c<<<n_blocks_elem, n_threads>>>(d_k1, d_elem_s1, d_elem_s2, d_elem_s3,
                                                  d_left_elem, d_right_elem);
        }

        rk_tempstorage<<<n_blocks_rk, n_threads>>>(d_c, d_k1, d_k1, 0.5);
        hipDeviceSynchronize();
        checkCudaError("error after stage 1.");

        // stage 2
        eval_surface<<<n_blocks_sides, n_threads>>>
                        (d_k1, d_rhs_surface_left, d_rhs_surface_right, 
                         d_s_length, 
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         d_left_elem, d_right_elem,
                         d_left_side_number, d_right_side_number,
                         d_Nx, d_Ny, t + 0.5*dt);

        eval_volume<<<n_blocks_elem, n_threads>>>
                        (d_k1, d_rhs_volume, 
                         d_xr, d_yr, d_xs, d_ys,
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         t + 0.5*dt);
        hipDeviceSynchronize();

        eval_rhs<<<n_blocks_elem, n_threads>>>(d_k1, d_rhs_volume, d_rhs_surface_left, d_rhs_surface_right,
                                              d_elem_s1, d_elem_s2, d_elem_s3, 
                                              d_left_elem, d_J, dt);
        hipDeviceSynchronize();

        if (limiter) {
            limit_c<<<n_blocks_elem, n_threads>>>(d_k1, d_elem_s1, d_elem_s2, d_elem_s3,
                                                  d_left_elem, d_right_elem);
        }
        checkCudaError("error after stage 2.");

        // final stage
        rk2<<<n_blocks_rk, n_threads>>>(d_c, d_k1);
        hipDeviceSynchronize();

        // check the convergence
        if (convergence && timestep > 0) {
            check_convergence<<<n_blocks_rk, n_threads>>>(d_c_prev, d_c);
            hipMemcpy(c, d_c_prev, local_num_elem * local_N * sizeof(double), hipMemcpyDeviceToHost);

            conv = c[0];
            for (i = 1; i < local_num_elem * local_N; i++) {
                if (c[i] > conv) {
                    conv = c[i];
                }
            }
        }
        if (convergence) {
            hipMemcpy(d_c_prev, d_c, local_num_elem * local_n_p * local_N * sizeof(double), hipMemcpyDeviceToDevice);
        }

        hipDeviceSynchronize();
        checkCudaError("error after final stage.");

    }

    printf("\n");
    free(max_lambda);
    //free(c);
    return t;
}
